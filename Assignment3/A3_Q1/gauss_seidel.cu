#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include<sys/time.h>

#define TOL 1e-5
#define ITER_LIMIT 1000
#define NUM_THREADS_PER_BLOCK 32

__managed__ float diff = 0.0;
__managed__ int nthreads, n;
__device__ int count = 0;
__managed__ int iterations = 0;
__device__ volatile int barrier_flag = 0;

__global__ void init(unsigned int seed, hiprandState_t* states) {
    hiprand_init(seed, blockIdx.x, 0, &states[blockIdx.x]);
}

__global__ void init_kernel(float*A, int span, hiprandState_t* states) {    
    int id  = threadIdx.x + blockIdx.x*blockDim.x;
    int val = ((n+2)*(n+2) < span*(id+1)) ? (n+2)*(n+2) : span*(id+1);
    for (int i = span*id; i < val; i++) {
        A[i] = (hiprand(&states[i])%100) / 100.0;
        // A[i] = i / (n+2);
    }
}

__global__ void gauss_seidel_kernel(float*A, int span){
    int id = threadIdx.x + blockIdx.x*blockDim.x;
    int val = ((n+2)*(n+2) < span*(id+1)) ? (n+2)*(n+2) : span*(id+1);
    int done = 0, row, col;
    float local_diff, temp;
    int local_sense = 0, last_count;
    __shared__ float local_area2[NUM_THREADS_PER_BLOCK/32];


    while(!done){
        // printf("Started function!\n");
        if(id == 0){
            diff = 0.0;
        }
        local_diff = 0.0;
        
        /************ Barrier ************/
        local_sense = (local_sense ? 0 : 1);
        __syncthreads();
        last_count = atomicAdd(&count, 1);

        if (last_count == nthreads - 1) {
            count = 0;
            barrier_flag = local_sense;
        }
        while (barrier_flag != local_sense);
        
        for(int i = span*id; i < val; i++){
            row = i/(n+2);
            col = i-row*(n+2);

            if (row != 0 && row != n+1 && col != 0 and col != n+1) {
                temp = A[i];
                A[i] = 0.2*(A[i] + A[i-1] + A[i+1] + A[i + n + 2] + A[i - n - 2]);
                local_diff += fabs(A[i] - temp); 
            }
        }

        unsigned mask = 0xffffffff;
        for (int i = warpSize/2; i > 0; i = i/2) {
            local_diff += __shfl_down_sync(mask, local_diff, i);
        }

        if(threadIdx.x % warpSize == 0) {
            local_area2[threadIdx.x/warpSize] = local_diff;
        }
        __syncthreads();

        if((threadIdx.x/(NUM_THREADS_PER_BLOCK/32)) == 0){
            local_diff = local_area2[threadIdx.x];
            for(int i=NUM_THREADS_PER_BLOCK/64; i>0; i=i/2){
                local_diff+= __shfl_down_sync(mask, local_diff, i);
            }
            if(threadIdx.x ==0){
                atomicAdd(&diff, local_diff);
            }

        }

        // if (id == 0) {
        //     atomicAdd(&diff, local_diff);
        // }
        
        /************ Barrier ************/
        local_sense = (local_sense ? 0 : 1);
        __syncthreads();
        last_count = atomicAdd(&count, 1);
        if (last_count == nthreads - 1) {
            count = 0;
            barrier_flag = local_sense;
        }
        while (barrier_flag != local_sense);

        iterations++;
        if ((diff/(n*n)< TOL) || (iterations == ITER_LIMIT)){
            done = 1;
        }

        /************ Barrier ************/
        local_sense = (local_sense ? 0 : 1);
        __syncthreads();
        last_count = atomicAdd(&count, 1);
        if (last_count == nthreads - 1) {
            count = 0;
            barrier_flag = local_sense;
        }
        while (barrier_flag != local_sense);
    }
}

int main(int argc, char*argv[]){
    float*A;
    struct timeval tv0, tv2;
    struct timezone tz0, tz2;
    hipError_t err;

    if(argc!=3){
        printf("Need dimensions of grid and number of threads\n");
        exit(1);
    }
    n = atoi(argv[1]);
    nthreads = atoi(argv[2]);

    hipMallocManaged((void**)&A, sizeof(float)*(n+2)*(n+2));

    int device = -1;
    hipGetDevice(&device);
	hipMemAdvise(A, sizeof(float)*(n+2)*(n+2), hipMemAdviseSetPreferredLocation, device);

    hiprandState_t* states;

    hipMalloc((void**) &states, (n+2) * (n+2) * sizeof(hiprandState_t));

    init<<<(n+2)*(n+2), 1>>>(time(0), states);
    err = hipGetLastError();
    if ( err != hipSuccess ) {
      		printf("CUDA ErrorINIT: %s\n", hipGetErrorString(err));
      		exit(-1);
   	}
    hipDeviceSynchronize();

    err = hipGetLastError();
    if ( err != hipSuccess ) {
      		printf("CUDA ErrorSYNC: %s\n", hipGetErrorString(err));
      		exit(-1);
    }

    unsigned long span = (n+2)*(n+2)/nthreads;
    if (span*nthreads < (n+2)*(n+2)){
        span++;
    }
    init_kernel<<< nthreads/NUM_THREADS_PER_BLOCK, NUM_THREADS_PER_BLOCK >>>(A, span, states);
    err = hipGetLastError();
    if ( err != hipSuccess ) {
      		printf("CUDA Error1: %s\n", hipGetErrorString(err));
      		exit(-1);
   	}
    hipDeviceSynchronize();
    err = hipGetLastError();
    if ( err != hipSuccess ) {
      		printf("CUDA Error2: %s\n", hipGetErrorString(err));
      		exit(-1);
   	}
    printf("Matrix initalization done!\n");

    gettimeofday(&tv0, &tz0);
    printf("Calling kernel!\n");
    gauss_seidel_kernel<<< nthreads/NUM_THREADS_PER_BLOCK, NUM_THREADS_PER_BLOCK >>>(A, span);
    err = hipGetLastError();
    if ( err != hipSuccess ) {
      		printf("CUDA Error3: %s\n", hipGetErrorString(err));
      		exit(-1);
   	}
    hipDeviceSynchronize();
    err = hipGetLastError();        // Get error code

    if ( err != hipSuccess ) {
            printf("CUDA Error4: %s\n", hipGetErrorString(err));
            exit(-1);
    }

    gettimeofday(&tv2, &tz2);
    printf("Time: %ld microseconds diff : %f iterations: %d\n", (tv2.tv_sec-tv0.tv_sec)*1000000+(tv2.tv_usec-tv0.tv_usec), diff / (n*n), iterations);
}
