#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include<sys/time.h>

#define TOL 1e-5
#define ITER_LIMIT 1000
#define NUM_THREADS_PER_BLOCK 32

__managed__ float diff = 0.0;
__managed__ int nthreads, n;
__device__ int count = 0;
__managed__ int iterations = 0;
__device__ volatile int barrier_flag = 0;

#ifdef CUDA_RANDOM
__global__ void init(unsigned int seed, hiprandState_t* states) {
    hiprand_init(seed, blockIdx.x, 0, &states[blockIdx.x]);
}

__global__ void init_kernel(float*A, int span, hiprandState_t* states) {    
    int id  = threadIdx.x + blockIdx.x*blockDim.x;
    int val = ((n+2)*(n+2) < span*(id+1)) ? (n+2)*(n+2) : span*(id+1);
    for (int i = span*id; i < val; i++) {
        A[i] = (hiprand(&states[i])%100) / 100.0;
        // A[i] = i / (n+2);
    }
}
#endif

#ifdef FIX
__global__ void init_sample_kernel(float*A, int span){
    int id  = threadIdx.x + blockIdx.x*blockDim.x;
    int val = ((n+2)*(n+2) < span*(id+1)) ? (n+2)*(n+2) : span*(id+1);
    for (int i = span*id; i < val; i++) {
        // A[i] = (hiprand(&states[i])%100) / 100.0;
        A[i] = float(i%100)/100;
    }

}
#endif


// call with A and nthreads
__global__ void gauss_seidel_kernel_general(float*A, int span){
    int id = threadIdx.x + blockIdx.x*blockDim.x;
    int val =  n*n;
    int done = 0, row, col;
    float local_diff, temp;
    int local_sense = 0, last_count, index, iterations = 0;
    __shared__ float local_area2[NUM_THREADS_PER_BLOCK/32];

    while(!done){
        // printf("Started function!\n");
        if(id == 0){
            diff = 0.0;
        }
        local_diff = 0.0;
        
        /************ Barrier ************/
        local_sense = (local_sense ? 0 : 1);
        __syncthreads();
        last_count = atomicAdd(&count, 1);

        if (last_count == nthreads - 1) {
            count = 0;
            barrier_flag = local_sense;
        }
        while (barrier_flag != local_sense);
        
        for(int i = id; i < val; i+=span){
            row = i/(n);
            col = i-row*(n);

            index = (row+1)*(n+2)+(col+1);

            temp = A[index];
            A[index] = 0.2*(A[index+1]+A[index-1]+A[index+n+2]+A[index-n-2]+A[index]);
            local_diff += fabs(A[index]-temp);
        }

        unsigned mask = 0xffffffff;
        for (int i = warpSize/2; i > 0; i = i/2) {
            local_diff += __shfl_down_sync(mask, local_diff, i);
        }

        if(threadIdx.x % warpSize == 0) {
            local_area2[threadIdx.x/warpSize] = local_diff;
        }
        __syncthreads();

        if((threadIdx.x/(NUM_THREADS_PER_BLOCK/32)) == 0){
            local_diff = local_area2[threadIdx.x];
            for(int i=NUM_THREADS_PER_BLOCK/64; i>0; i=i/2){
                local_diff+= __shfl_down_sync(mask, local_diff, i);
            }
            if(threadIdx.x ==0){
                atomicAdd(&diff, local_diff);
            }

        }
        /************ Barrier ************/
        local_sense = (local_sense ? 0 : 1);
        __syncthreads();
        last_count = atomicAdd(&count, 1);
        if (last_count == nthreads - 1) {
            count = 0;
            barrier_flag = local_sense;
        }
        while (barrier_flag != local_sense);

        iterations++;
        if ((diff/(n*n)< TOL) || (iterations == ITER_LIMIT)){
            done = 1;
        }

        /************ Barrier ************/
        local_sense = (local_sense ? 0 : 1);
        __syncthreads();
        last_count = atomicAdd(&count, 1);
        if (last_count == nthreads - 1) {
            count = 0;
            barrier_flag = local_sense;
        }
        while (barrier_flag != local_sense);
    }
    if(id == 0){
        printf("Iterations : %d\n", iterations);
    }
}


__global__ void gauss_seidel_kernel(float*A, int span){
    int id = threadIdx.x + blockIdx.x*blockDim.x;
    int val = ((n+2)*(n+2) < span*(id+1)) ? (n+2)*(n+2) : span*(id+1);
    int done = 0, row, col;
    float local_diff, temp;
    int local_sense = 0, last_count;
    __shared__ float local_area2[NUM_THREADS_PER_BLOCK/32];


    while(!done){
        // printf("Started function!\n");
        if(id == 0){
            diff = 0.0;
        }
        local_diff = 0.0;
        
        /************ Barrier ************/
        local_sense = (local_sense ? 0 : 1);
        __syncthreads();
        last_count = atomicAdd(&count, 1);

        if (last_count == nthreads - 1) {
            count = 0;
            barrier_flag = local_sense;
        }
        while (barrier_flag != local_sense);
        
        for(int i = span*id; i < val; i++){
            row = i/(n+2);
            col = i-row*(n+2);

            if (row != 0 && row != n+1 && col != 0 and col != n+1) {
                temp = A[i];
                A[i] = 0.2*(A[i] + A[i-1] + A[i+1] + A[i + n + 2] + A[i - n - 2]);
                local_diff += fabs(A[i] - temp); 
            }
        }

        unsigned mask = 0xffffffff;
        for (int i = warpSize/2; i > 0; i = i/2) {
            local_diff += __shfl_down_sync(mask, local_diff, i);
        }

        if(threadIdx.x % warpSize == 0) {
            local_area2[threadIdx.x/warpSize] = local_diff;
        }
        __syncthreads();

        if((threadIdx.x/(NUM_THREADS_PER_BLOCK/32)) == 0){
            local_diff = local_area2[threadIdx.x];
            for(int i=NUM_THREADS_PER_BLOCK/64; i>0; i=i/2){
                local_diff+= __shfl_down_sync(mask, local_diff, i);
            }
            if(threadIdx.x ==0){
                atomicAdd(&diff, local_diff);
            }

        }

        // if (id == 0) {
        //     atomicAdd(&diff, local_diff);
        // }
        
        /************ Barrier ************/
        local_sense = (local_sense ? 0 : 1);
        __syncthreads();
        last_count = atomicAdd(&count, 1);
        if (last_count == nthreads - 1) {
            count = 0;
            barrier_flag = local_sense;
        }
        while (barrier_flag != local_sense);

        iterations++;
        if ((diff/(n*n)< TOL) || (iterations == ITER_LIMIT)){
            done = 1;
        }

        /************ Barrier ************/
        local_sense = (local_sense ? 0 : 1);
        __syncthreads();
        last_count = atomicAdd(&count, 1);
        if (last_count == nthreads - 1) {
            count = 0;
            barrier_flag = local_sense;
        }
        while (barrier_flag != local_sense);
    }
}

int main(int argc, char*argv[]){
    float*A;
    struct timeval tv0, tv2;
    struct timezone tz0, tz2;
    hipError_t err;

    if(argc!=3){
        printf("Need dimensions of grid and number of threads\n");
        exit(1);
    }
    n = atoi(argv[1]);
    nthreads = atoi(argv[2]);

    hipMallocManaged((void**)&A, sizeof(float)*(n+2)*(n+2));

    int device = -1;
    hipGetDevice(&device);
	hipMemAdvise(A, sizeof(float)*(n+2)*(n+2), hipMemAdviseSetPreferredLocation, device);

    unsigned long span = (n+2)*(n+2)/nthreads;
    if (span*nthreads < (n+2)*(n+2)){
        span++;
    }

    #ifdef FIX
    init_sample_kernel<<< nthreads/NUM_THREADS_PER_BLOCK, NUM_THREADS_PER_BLOCK >>>(A,span);
    hipDeviceSynchronize();
    err = hipGetLastError();
    if ( err != hipSuccess ) {
      		printf("CUDA Error2: %s\n", hipGetErrorString(err));
      		exit(-1);
   	}
    #endif

    #ifdef CUDA_RANDOM

    hiprandState_t* states;

    hipMalloc((void**) &states, (n+2) * (n+2) * sizeof(hiprandState_t));

    init<<<(n+2)*(n+2), 1>>>(time(0), states);
    err = hipGetLastError();
    if ( err != hipSuccess ) {
      		printf("CUDA ErrorINIT: %s\n", hipGetErrorString(err));
      		exit(-1);
   	}
    hipDeviceSynchronize();

    err = hipGetLastError();
    if ( err != hipSuccess ) {
      		printf("CUDA ErrorSYNC: %s\n", hipGetErrorString(err));
      		exit(-1);
    }

    
    init_kernel<<< nthreads/NUM_THREADS_PER_BLOCK, NUM_THREADS_PER_BLOCK >>>(A, span, states);
    err = hipGetLastError();
    if ( err != hipSuccess ) {
      		printf("CUDA Error1: %s\n", hipGetErrorString(err));
      		exit(-1);
   	}
    hipDeviceSynchronize();
    err = hipGetLastError();
    if ( err != hipSuccess ) {
      		printf("CUDA Error2: %s\n", hipGetErrorString(err));
      		exit(-1);

   	}
    #endif
    printf("Matrix initalization done!\n");

    gettimeofday(&tv0, &tz0);
    printf("Calling kernel!\n");
    gauss_seidel_kernel<<< nthreads/NUM_THREADS_PER_BLOCK, NUM_THREADS_PER_BLOCK >>>(A, span);
    err = hipGetLastError();
    if ( err != hipSuccess ) {
      		printf("CUDA Error3: %s\n", hipGetErrorString(err));
      		exit(-1);
   	}
    hipDeviceSynchronize();
    err = hipGetLastError();        // Get error code

    if ( err != hipSuccess ) {
            printf("CUDA Error4: %s\n", hipGetErrorString(err));
            exit(-1);
    }

    gettimeofday(&tv2, &tz2);
    printf("Time: %ld microseconds diff : %f iterations: %d\n", (tv2.tv_sec-tv0.tv_sec)*1000000+(tv2.tv_usec-tv0.tv_usec), diff / (n*n), iterations);
}
